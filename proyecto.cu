
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <math.h>
#include <vector>

#include <time.h>

#define ancho 1280
#define alto 720
#define totalPixeles ancho*alto

#define totalhilos 32

typedef int tamPixel;

using namespace std;

void llenarVectores(tamPixel *V){

        for(int i=0;i<totalPixeles;i++){
                srand(time(NULL));
                V[i]=rand()%256;
        }
}

__global__ void convertirRGBtoYCoCg(tamPixel *RY,tamPixel *GCg, tamPixel *BCo, int height, int width){
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;
	int index = Row*width+Col;
	
	if ((Row < height) && (Col < width)) {
		RY[index] = ((1/4)*RY[index]) + ((1/2)*GCg[index]) + ((1/4)*BCo[index]);
		GCg[index] = ((-1/4)*RY[index]) + ((1/2)*GCg[index]) + ((-1/4)*BCo[index]);
		BCo[index] = ((1/2)*RY[index]) + ((-1/2)*BCo[index]);
	}
}

__global__ void suma(tamPixel *RY,tamPixel *GCg, tamPixel *BCo, int height, int width){
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;
	int index = Row*width+Col;
	
	if ((Row < height) && (Col < width)) {
		BCo[index] = RY[index]+GCg[index];
	}
}

int main(){

		tamPixel *R,*G,*B;
		tamPixel *RY, *GCg, *BCo;

		int size = totalPixeles * sizeof(tamPixel);

		R = (tamPixel *)malloc(size);
		G = (tamPixel *)malloc(size); 
		B = (tamPixel *)malloc(size);		

        llenarVectores(R);
        llenarVectores(G);
        llenarVectores(B);

        hipMalloc((void **)&RY, size);
		hipMalloc((void **)&GCg, size);
		hipMalloc((void **)&BCo, size);

        hipMemcpy(RY,R,size,hipMemcpyHostToDevice);
		hipMemcpy(GCg,G,size,hipMemcpyHostToDevice);
		hipMemcpy(BCo,B,size,hipMemcpyHostToDevice);

		dim3 DimGrid(((ancho-1)/totalhilos)+1, ((alto-1)/totalhilos)+1, 1);//ver
		dim3 DimBlock(totalhilos, totalhilos, 1);

		convertirRGBtoYCoCg<<<DimGrid,DimBlock>>>(RY,GCg,BCo, alto, ancho);

        cout<<"todo bn"<<endl;

}
