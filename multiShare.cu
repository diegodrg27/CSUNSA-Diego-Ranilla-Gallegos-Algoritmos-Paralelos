
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <time.h> 
#include <stdio.h>
#define ROW 15 //filas Matriz 1
#define COL 15 // columna Matriz 2

#define N ROW*COL // Cantidad de elementos en Matriz 3

#define THREADS 5
using namespace std;

void createMatrixHost(float**& host, int row, int col, int size){
    host = (float **)malloc(row*sizeof(float*));
    host[0]=(float *)malloc(size);
	
    for (int i=1; i<row;++i){
        host[i]=host[i-1]+col;
    }

}

void createMatrixHostCUDA(float**& host, float**& device, float **& aux, int row, int col, int size){
    host = (float **)malloc(row*sizeof(float*));
    host[0]=(float *)malloc(size);
    aux =(float **)malloc(row*sizeof(float*));

    hipMalloc((void **)&aux[0],size);
    hipMalloc((void **)&device,row*sizeof(float*));

    for (int i=1; i<row;++i){
        host[i]=host[i-1]+col;
        aux[i]=aux[i-1]+col;
    }
    hipMemcpy(device, aux, row*sizeof(float*), hipMemcpyHostToDevice);
}

void Multiplicacion(float** A, float** B, float** P){
        for(int i=0;i<ROW;i++){
                for(int j=0;j<COL;j++){
                        float Sum=0.0;
                        for(int k=0;k<COL;k++){
                                Sum += A[i][k]*B[k][j];
                        }
                        P[i][j] = Sum;
                }
        }
}

__global__ void MatrixMulKernel2(float** A, float** B, float** P)
{
        __shared__ float A_b[THREADS][THREADS];
        __shared__ float B_b[THREADS][THREADS];
        __shared__ float B_b2[THREADS][THREADS];

        __shared__ float R_b[THREADS][THREADS];
	__shared__ float R_b2[THREADS][THREADS];

        int Row = blockIdx.y * THREADS + threadIdx.y;
        int Col = blockIdx.x * THREADS * 2 + threadIdx.x;

	R_b[threadIdx.y][threadIdx.x] = 0.0;
	R_b2[threadIdx.y][threadIdx.x] = 0.0;
        __syncthreads(); 

	for(int i = 0;i < ceil(COL/(float)THREADS);i++){
		A_b[threadIdx.y][threadIdx.x] = 0.0;
		B_b[threadIdx.y][threadIdx.x] = 0.0;
		B_b2[threadIdx.y][threadIdx.x] = 0.0;
		__syncthreads();

		if ((Row<ROW) && (i*THREADS + threadIdx.x<COL)){
	                A_b[threadIdx.y][threadIdx.x] = A[Row] [i*THREADS + threadIdx.x];
			
		}

		if ((i*THREADS + threadIdx.y<COL) && (Col<COL)){
	                B_b[threadIdx.y][threadIdx.x] = B[i*THREADS + threadIdx.y][Col]; 
		}

		if ((i*THREADS + threadIdx.y<COL) && (Col+THREADS<COL)){
			B_b2[threadIdx.y][threadIdx.x] = B[i*THREADS + threadIdx.y][Col+THREADS]; 
		}

                __syncthreads();

                for (int k = 0; k < THREADS; k++) {
                        R_b[threadIdx.y][threadIdx.x] += A_b[threadIdx.y][k] * B_b[k][threadIdx.x];
			R_b2[threadIdx.y][threadIdx.x] += A_b[threadIdx.y][k] * B_b2[k][threadIdx.x];
                }
                __syncthreads();
	 }

	 if((Row<ROW) && (Col<COL)){
	         P[Row][Col] = R_b[threadIdx.y][threadIdx.x];
	}

	 if((Row<ROW) && (Col+THREADS<COL)){
  		 P[Row][Col+THREADS] = R_b2[threadIdx.y][threadIdx.x];
	}
}

void llenarVector(float **V, int row, int col){
    for(int i=0;i<row;i++){
	for(int j=0;j<col;j++){
	        V[i][j]=rand()%11;
	}
    }
}

void imprimir(float **M, int row, int col){
        for(int i=0;i<row;i++){
                for(int j=0;j<col;j++){
                        cout<<M[i][j]<<" ";
                }
                cout<<endl;
        }
        cout<<endl;
}


int main(){
	float **a, **b, **c3,**c2;
	//////////////////////////////////////////
	float **d_a, **d_b, **d_c3;
	float **a_aux, **b_aux, **c_aux3;
	///////////////////////////////////////////

	int size = N * sizeof(float*);
	
	dim3 DimGrid(ceil((((COL-1)/(float)THREADS)+1)/2), ((ROW-1)/THREADS)+1, 1);
      	dim3 DimBlock(THREADS, THREADS, 1);
	createMatrixHostCUDA(a,d_a,a_aux,ROW,COL,size);
	createMatrixHostCUDA(b,d_b,b_aux,ROW,COL,size);
	
	createMatrixHostCUDA(c3,d_c3,c_aux3,ROW,COL,size);

	createMatrixHost(c2,ROW,COL,size);
	
    	llenarVector(a,ROW,COL);
	llenarVector(b,ROW,COL);

	imprimir(a,ROW,COL);
	imprimir(b,ROW,COL);

	Multiplicacion(a,b,c2);
	imprimir(c2,ROW,COL);

	hipMemcpy(a_aux[0], a[0], size, hipMemcpyHostToDevice);
	hipMemcpy(b_aux[0], b[0], size, hipMemcpyHostToDevice);
	MatrixMulKernel2<<<DimGrid,DimBlock>>>(d_a,d_b,d_c3);
	
	hipMemcpy(c3[0],c_aux3[0], size, hipMemcpyDeviceToHost);
	imprimir(c3,ROW,COL);
}
