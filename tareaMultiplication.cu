
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#define WIDTH 448
#define N WIDTH*WIDTH
#define totalhilos 64
#define TILE_WIDTH totalhilos
using namespace std;


__global__ void MatrixMulKernel(float* A, float* B, float* P)
{
        int Row = blockIdx.y*blockDim.y +threadIdx.y;
        int Col = blockIdx.x*blockDim.x +threadIdx.x;

        if((Row < WIDTH) && (Col < WIDTH)){
                float Pvalue = 0.0;

                for(int k=0;k<WIDTH;k++){
                        Pvalue+= A[Row*WIDTH+k] * B[k*WIDTH+Col];
                }
                P[Row*WIDTH+Col] = Pvalue;
        }
}

__global__ void MatrixMulTiledKernel(float* d_M, float* d_N, float* d_P) {
        __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
        __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

        int bx = blockIdx.x; int by = blockIdx.y;
        int tx = threadIdx.x; int ty = threadIdx.y;

        int Row = by * TILE_WIDTH + ty;
        int Col = bx * TILE_WIDTH + tx;

    float Pvalue = 0;

    for (int ph = 0; ph < WIDTH/TILE_WIDTH; ++ph) {
                Mds[ty][tx] = d_M[Row*WIDTH + ph*TILE_WIDTH + tx];
                Nds[ty][tx] = d_N[(ph*TILE_WIDTH + ty)*WIDTH + Col];
                __syncthreads();

                for (int k = 0; k < TILE_WIDTH; ++k) {
                        Pvalue += Mds[ty][k] * Nds[k][tx];
                }
                __syncthreads();
        }
        d_P[Row*WIDTH + Col] = Pvalue;
}


void llenarVector(float *V){
    for(int i=0;i<N;i++){
        V[i]=rand()%11;
    }
}

void imprimir(float *M){
        for(int i=0;i<WIDTH;i++){
                for(int j=0;j<WIDTH;j++){
                        cout<<M[i*WIDTH+j]<<" ";
                }
                cout<<endl;
        }
        cout<<endl;
}


int main(){
        cout<<"inicio"<<endl;
        float *d_A, *d_B, *d_C;
        float *dd_A, *dd_B, *dd_C;

        float h_A[N], h_B[N], h_C[N], hh_C[N];

         hipEvent_t start;
        hipEvent_t stop;

        hipEvent_t start2;
        hipEvent_t stop2;

        llenarVector(h_A);
        llenarVector(h_B);
        //llenarVector(h_C);

//      imprimir(h_A);
//      imprimir(h_B);

        hipMalloc((void **)&d_A, N*sizeof(float));
        hipMalloc((void **)&d_B, N*sizeof(float));
        hipMalloc((void **)&d_C, N*sizeof(float));

        hipMalloc((void **)&dd_A, N*sizeof(float));
        hipMalloc((void **)&dd_B, N*sizeof(float));
         hipMalloc((void **)&dd_C, N*sizeof(float));

        hipMemcpy(d_A,h_A,N*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(d_B,h_B,N*sizeof(float),hipMemcpyHostToDevice);

        hipMemcpy(dd_A,h_A,N*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(dd_B,h_B,N*sizeof(float),hipMemcpyHostToDevice);

        dim3 DimGrid(((WIDTH-1)/totalhilos)+1, ((WIDTH-1)/totalhilos)+1, 1);//ver
        dim3 DimBlock(totalhilos, totalhilos, 1);

        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start,0);
        MatrixMulKernel<<<DimGrid,DimBlock>>>(d_A,d_B,d_C);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        float elapsedTime;

        hipEventElapsedTime(&elapsedTime,start,stop);
        cout<<"Tiempo de ejecucion Multiplicacion Normal: "<<elapsedTime<<endl;
        hipEventDestroy(start);
        hipEventDestroy(stop);


        /*cudaEventCreate(&start2);
        cudaEventCreate(&stop2);
        cudaEventRecord(start2,0);
        MatrixMulTiledKernel<<<DimGrid,DimBlock>>>(dd_A,dd_B,dd_C);
        cudaEventRecord(stop2,0);
        cudaEventSynchronize(stop2);
        float elapsedTime2;
        cudaEventElapsedTime(&elapsedTime2,start2,stop2);
        cout<<"Tiempo de ejecucion Multiplicacion Tiled: "<<elapsedTime2<<endl;
        cudaEventDestroy(start2);
        cudaEventDestroy(stop2); */
           hipMemcpy(h_C,d_C,N*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(hh_C,dd_C,N*sizeof(float),hipMemcpyDeviceToHost);
        cout<<"Multiplicación tradicional"<<endl;
//      imprimir(h_C);

        cout<<endl;
        cout<<"Multiplicacion Tiled"<<endl;
//      imprimir(hh_C);


}
